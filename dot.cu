/*
 * dot.cu
 * Author: Simon Rüegg
 *
 * includes setup funtion and kernel function called from "driver" program
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>

#define BLOCK_SIZE 1024

__global__ void cu_calcDot(unsigned long long *vector_d, uint32_t vectorSize, unsigned long long *dotResult_d)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= vectorSize) return;

	unsigned long long force_d = x < vectorSize/2 ? x+1 : vectorSize/2 - (x-vectorSize/2);
	unsigned long long distance_d = (x % 10) + 1;

	*dotResult_d = 0;

    // calculate products
	vector_d[x] = force_d * distance_d;
}

__global__ void cu_reduce(unsigned long long *vector_d, uint32_t vectorSize, unsigned long long *result_d)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(x >= vectorSize) return;
	
	// reduce to sum
	atomicAdd(result_d, vector_d[x]);
}

// This function is called from the host computer.
// It manages memory and calls the function that is executed on the GPU
extern "C" void calcDot(uint32_t vectorSize, unsigned long long *dotResult)
{
    // force_d and dotResult reside on the GPU
	unsigned long long *force_d;
	unsigned long long *dotResult_d;
	hipError_t result;

    // allocate space on the device
	result = hipMalloc((void **)&force_d, sizeof(unsigned long long) * vectorSize);
	if (result != hipSuccess)
	{
		fprintf(stderr, "hipMalloc (force) failed: %s\n", hipGetErrorString(result));
		exit(1);
	}
	result = hipMalloc((void **)&dotResult_d, sizeof(unsigned long long));
	if (result != hipSuccess)
	{
		fprintf(stderr, "hipMalloc (dotResult) failed: %s\n", hipGetErrorString(result));
		exit(1);
	}

    // set execution configuration
	dim3 dimblock(BLOCK_SIZE);
	dim3 dimgrid(ceil((double)vectorSize / BLOCK_SIZE));

	// Start timing
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);

    // actual computation: Call the kernel
	cu_calcDot<<<dimgrid, dimblock>>>(force_d, vectorSize, dotResult_d);
	cu_reduce<<<dimgrid, dimblock>>>(force_d, vectorSize, dotResult_d);

    // transfer results back to host
	result = hipMemcpy(dotResult, dotResult_d, sizeof(unsigned long long), hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy host <- dev (dotResult) failed: %s\n", hipGetErrorString(result));
		exit(1);
	}

	// Stop timing
    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);
    printf("Time elapsed: %ld.%06lds\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

    // release the memory on the GPU
	result = hipFree(force_d);
	if (result != hipSuccess)
	{
		fprintf(stderr, "hipFree (force) failed: %s\n", hipGetErrorString(result));
		exit(1);
	}
	result = hipFree(dotResult_d);
	if (result != hipSuccess)
	{
		fprintf(stderr, "hipFree (dotResult) failed: %s\n", hipGetErrorString(result));
		exit(1);
	}
}
